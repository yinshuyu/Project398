#include "hip/hip_runtime.h"
// Utility and system includes
#include <hip/hip_runtime_api.h>
#include <math.h>
#include "md5.h"
#include "BruteForceAttack_loc.h"

#define BLOCK_SIZE 32
typedef unsigned int uint;

// n^k possible strings of length k that can be formed from a set of n characters
__global__ void Kernel_BruteForce(
	const char* __restrict__ hash,
	char* result,
	bool* found,
	unsigned msgLgth,
	unsigned startIndex,
	unsigned endIndex
)
{
	unsigned bx = blockIdx.x;
	unsigned by = blockIdx.y;
	unsigned tx = threadIdx.x;
	unsigned ty = threadIdx.y;
	unsigned col = bx * blockDim.x + tx;
	unsigned row = by * blockDim.y + ty;

	unsigned index = startIndex + (row * blockDim.x + col);

	if (index < endIndex)
	{
		char* test = new char[msgLgth];

		// get the test msg for this thread
		{
			unsigned dividend = index;
			unsigned quotient = 0;
			//unsigned remainder = 0;

			unsigned i = 0;
			for (unsigned divisor = endIndex/RANGE_0; i < msgLgth - 1; i++, divisor /= RANGE_0)
			{
				quotient = floor((float)(dividend) / (float)divisor);
				//	remainder = dividend % divisor;
				dividend %= divisor;

				test[i] = START_0 + quotient;

				//	dividend = remainder;
			}

			test[i] = START_0 + dividend;

			
		}

		__syncthreads();

		char chash[MD5_STRING_SIZE];

		//hash a single dictionary password to MD5hash format
		md5::md5_t md5;
		md5.process(test, msgLgth);
		md5.finish();
		md5.get_string(chash);

		__syncthreads();

		//compare if the dictionary MD5hash format is the same as the user input
		bool match = true;

		for (unsigned i = 0; i < MD5_STRING_SIZE; i++)
		{
			if (chash[i] != hash[i])
			{
				match = false;
				break;
			}
		}

		if (match)
		{
			*found = true;

			//memcpy(result, &(list[size * msgMaxLgth]), length);
			for (unsigned i = 0; i < msgLgth; i++)
				result[i] = test[i];
		}

		__syncthreads();

		delete[] test;
	}

}


// A function to find the factorial.
int possibleAmount(int n, int k)
{
	int result = 1;

	for (int i = 0; i < k; i++)
		result *= n;

	return result;
}

//Used in GPU mode
//Calls kernel function
extern "C" void GPUBruteForce(
	char* hash,		//user input hash
	char* result,	//password result found by kernel
	bool* found,

	unsigned msgMinLgth,	//the min search length
	unsigned msgMaxLgth,	//the max search length

	unsigned tileSize
)
{
	unsigned maxTileSize = tileSize * tileSize;
	//printf("%d\n\n", maxTileSize);

	for (unsigned msgLgth = msgMinLgth; msgLgth <= msgMaxLgth; msgLgth++)
	{
		unsigned endIndex = possibleAmount(RANGE_0, msgLgth);
		//printf("%d\n\n", endIndex);

		for (unsigned startIndex = 0; startIndex < endIndex; startIndex += maxTileSize)
		{
			unsigned remainderIndex = endIndex - startIndex;
			unsigned y_size = tileSize;

			if(maxTileSize > remainderIndex)
				y_size = ceil((float)remainderIndex / (float)tileSize);

			//printf("%d\n", startIndex);
			//printf("%d\n", y_size);

			dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE, 1); //32 by 32

			dim3 gridDim(ceil((float)(tileSize) / (float)BLOCK_SIZE), ceil((float)(y_size) / (float)BLOCK_SIZE), 1);

			Kernel_BruteForce << <gridDim, blockDim >> > (
				hash,
				result,
				found,
				msgLgth,
				startIndex,
				endIndex);

			getLastCudaError("Kernel_BruteForce failed\n");

			hipDeviceSynchronize();

			/*if (found)
				return;*/
		}	
	}
}