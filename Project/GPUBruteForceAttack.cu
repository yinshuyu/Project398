#include "hip/hip_runtime.h"
/*Start Header
******************************************************************/
/*!
\file GPUBruteForceAttack.cu
\author	Yin shuyu, yin.s, 1802075
\par yin.s\@digipen.edu
\date Apr 19, 2021
\brief  CS398 Final Project
Copyright (C) 2021 DigiPen Institute of Technology.
Reproduction or disclosure of this file or its contents without the
prior written consent of DigiPen Institute of Technology is prohibited.
*/
/* End Header
*******************************************************************/

// Utility and system includes
#include <hip/hip_runtime_api.h>
#include <math.h>
#include "md5.h"
#include "BruteForceAttack_loc.h"

typedef unsigned int uint;

__global__ void Kernel_BruteForce(
	const char* __restrict__ hash,
	char* result,
	bool* found,
	unsigned msgLgth,
	unsigned startIndex,
	unsigned endIndex
)
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId * (blockDim.x * blockDim.y)
		+ (threadIdx.y * blockDim.x) + threadIdx.x;

	// thread index -- also the combination's index
	unsigned index = startIndex + threadId;

	if (index < endIndex)
	{
		char* test = new char[msgLgth];

		// get the test msg for this thread
		{
			unsigned dividend = index;
			unsigned quotient = 0;
			//unsigned remainder = 0;

			unsigned i = 0;
			for (unsigned divisor = endIndex/ RANGE; i < msgLgth - 1; i++, divisor /= RANGE)
			{
				quotient = floor((float)(dividend) / (float)divisor);
				//	remainder = dividend % divisor;
				dividend %= divisor;

				test[i] = START + quotient;

				//	dividend = remainder;
			}

			test[i] = START + dividend;

			
		}

		__syncthreads();

		char chash[MD5_STRING_SIZE];

		//hash a single dictionary password to MD5hash format
		md5::md5_t md5;
		md5.process(test, msgLgth);
		md5.finish();
		md5.get_string(chash);

		__syncthreads();

		//compare if the dictionary MD5hash format is the same as the user input
		bool match = true;

		for (unsigned i = 0; i < MD5_STRING_SIZE; i++)
		{
			if (chash[i] != hash[i])
			{
				match = false;
				break;
			}
		}

		if (match)
		{
			*found = true;

			//memcpy(result, &(list[size * msgMaxLgth]), length);
			for (unsigned i = 0; i < msgLgth; i++)
				result[i] = test[i];
		}

		__syncthreads();

		delete[] test;
	}

}


// n^k possible strings of length k that can be formed from a set of n characters
int possibleAmount(int n, int k)
{
	int result = 1;

	for (int i = 0; i < k; i++)
		result *= n;

	return result;
}

//Used in GPU mode
//Calls kernel function
extern "C" void GPUBruteForce(
	char* hash,		//user input hash
	char* result,	//password result found by kernel
	bool* found,

	unsigned msgMinLgth,	//the min search length
	unsigned msgMaxLgth,	//the max search length

	unsigned tileSize,
	unsigned blockSize
)
{
	unsigned maxTileSize = tileSize * tileSize;
	//printf("%d\n\n", maxTileSize);
	bool hashfound; // use to break when hash in find in one of the kenrel

	// split the all combination in different msg length batch 
	for (unsigned msgLgth = msgMinLgth; msgLgth <= msgMaxLgth; msgLgth++)
	{
		// number combination in this ASCII range and msg length
		unsigned endIndex = possibleAmount(RANGE, msgLgth);
		//printf("%d\n\n", endIndex);

		for (unsigned startIndex = 0; startIndex < endIndex; startIndex += maxTileSize)
		{
			unsigned remainderIndex = endIndex - startIndex;
			unsigned y_size = tileSize;

			if(maxTileSize > remainderIndex)
				y_size = ceil((float)remainderIndex / (float)tileSize);

			//printf("%d\n", startIndex);
			//printf("%d\n", y_size);

			dim3 blockDim(blockSize, blockSize, 1); //32 by 32

			dim3 gridDim(ceil((float)(tileSize) / (float)blockSize), ceil((float)(y_size) / (float)blockSize), 1);

			Kernel_BruteForce << <gridDim, blockDim >> > (
				hash,
				result,
				found,
				msgLgth,
				startIndex,
				endIndex);

			hipMemcpy(&hashfound, found, sizeof(bool), hipMemcpyDeviceToHost);

			getLastCudaError("Kernel_BruteForce failed\n");

			hipDeviceSynchronize();

			// stop if hash is found
			if (hashfound)
				return;
		}	
	}
}