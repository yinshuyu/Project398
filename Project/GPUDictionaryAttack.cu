#include "hip/hip_runtime.h"
/*Start Header
******************************************************************/
/*!
\file GPUDictionaryAttack.cu
\author Luo Yu Xuan, yuxuan.luo, 1802205
\par yuxuan.luo\@digipen.edu
\date Apr 19, 2021
\brief  CS398 Final Project
Copyright (C) 2021 DigiPen Institute of Technology.
Reproduction or disclosure of this file or its contents without the
prior written consent of DigiPen Institute of Technology is prohibited.
*/
/* End Header
*******************************************************************/
// Utility and system includes
#include <hip/hip_runtime_api.h>
#include "md5.h"
#include <math.h>


#define BLOCK_SIZE 32
typedef unsigned int uint;



__global__ void Kernel_ScanDictionary(
	const char* __restrict__ hash,
	char * list, 
	unsigned listSize,
	char* result, 
	unsigned msgMaxLgth,
	bool * resultfound
)
{

	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int col = bx * blockDim.x + tx;
	int row = by * blockDim.y + ty;

	int x_size = gridDim.x * blockDim.x;
	int size = row * x_size + col;

	if (size < listSize)
	{
		char chash[MD5_STRING_SIZE];

		unsigned length = 0;

		while (list[size *msgMaxLgth + length] != '\0')
			length++;


		//hash a single dictionary password to MD5hash format
		md5::md5_t md5;
		md5.process(&(list[size * msgMaxLgth]), length);
		md5.finish();
		md5.get_string(chash);

		//compare if the dictionary MD5hash format is the same as the user input
		int i = 0;
		bool match = true;
		while (i < MD5_STRING_SIZE)
		{
			if (chash[i] != hash[i])
			{
				match = false;
				break;
			}
			i++;
		}

		if (match)
		{
			//convert the dictionary char * to std::string
			//result = &(list[size * msgMaxLgth]);

			//memcpy(result, &(list[size * msgMaxLgth]), length);

			*resultfound = true;
			for (int i = 0; i < length; i++)
			{
				result[i] = list[size * msgMaxLgth + i];
			}
			//found the password in the dictionary
		}
	}

}




//Used in GPU mode
//Calls kernel function
extern "C" void GPUScanDictionary(
	char* hash,  //user input hash
	char* result,      //password result found by kernel

	char* list,			 //dictionary list
	unsigned listSize,	//size of dictionary
	unsigned msgMaxLgth, //the max length of a password in the dictionary

	unsigned tileSize,
	hipStream_t stream, //stream number, 
	bool * resultfound
)
{
	uint block = BLOCK_SIZE;

	dim3 blockDim(block, block, 1); //32 by 32

	int y_size = ceil((float)listSize / (float)tileSize);
	dim3 gridDim(ceil((float)(tileSize) / (float)block), ceil((float)(y_size) / (float)block), 1);

	Kernel_ScanDictionary << <gridDim, blockDim , 0, stream>> > (
		hash,
		list,
		listSize,
		result,
		msgMaxLgth, resultfound);

	getLastCudaError("Kernel_ScanDictionary failed\n");

	hipDeviceSynchronize();
}